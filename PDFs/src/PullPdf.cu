#include "hip/hip_runtime.h"
/*****************************************************************************/
// Author: Xuefeng Ding <xuefeng.ding.physics@gmail.com>
// Insitute: Gran Sasso Science Institute, L'Aquila, 67100, Italy
// Date: 2018 April 7th
// Version: v1.0
// Description: GooStats, a statistical analysis toolkit that runs on GPU.
//
// All rights reserved. 2018 copyrighted.
/*****************************************************************************/
#include "PullPdf.h"

#define M_PI_L 3.141592653589793238462643383279502884L
__host__ PullPdf::PullPdf(std::string n, Variable* var, fptype m,fptype s,fptype mt) :
  GooPdf(nullptr, n),
  index (registerParameter(var)),
  mean(m*mt),
  sigma(s*mt),
  masstime(mt)
{}


__host__ double PullPdf::calculateNLL () const {
  const double counts = masstime*host_params[index];
  const double constTerm(IsChisquareFit()?(0.5*log(2*M_PI_L*sigma*sigma)):0);
#ifdef NLL_CHECK
  printf("log(L) %.12le pull\n",(counts-mean)*(counts-mean)/(2*sigma*sigma)+constTerm);
#endif
  return (counts-mean)*(counts-mean)/(2*sigma*sigma)+constTerm;
}

